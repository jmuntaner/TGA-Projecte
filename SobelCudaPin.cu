#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/times.h>
#include <sys/resource.h>
#include <math.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define THREADS 16

char* filename;
const char* seq_out = "gradientSEQ.png";
const char* cuda_out = "gradientCUDA.png";

void sobelSeq(unsigned char *image, unsigned char *output, int width, int height) {
    float gx = 0;
    float gy = 0;
    float s = 0;
    for (int i = 1; i < height-1; ++i) {
        for (int j = 1; j < width-1; ++j) {
            gx = image[(i-1)*width+j-1]+2*image[i*width+j-1]+image[(i+1)*width+j-1]
            -image[(i-1)*width+j+1]-2*image[i*width+j+1]-image[(i+1)*width+j+1];
            gy = image[(i-1)*width+j-1]+2*image[(i-1)*width+j]+image[(i-1)*width+j+1]
            -image[(i+1)*width+j-1]-2*image[(i+1)*width+j]-image[(i+1)*width+j+1];
            s = (int) sqrt(gx*gx+gy*gy);
            s = s > 255 ? 255:s;
            output[i*width+j] = s;
        }
    }
}

__global__ void KernelSobelElement (unsigned char *image, unsigned char *output, int width, int height) {

  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  float gx = 0;
  float gy = 0;
  float s = 0;
  if(i > 0 && j > 0 && i < (height-1) && j < (width-1)) {
      gx = image[(i-1)*width+j-1]+2*image[i*width+j-1]+image[(i+1)*width+j-1]
      -image[(i-1)*width+j+1]-2*image[i*width+j+1]-image[(i+1)*width+j+1];
      gy = image[(i-1)*width+j-1]+2*image[(i-1)*width+j]+image[(i-1)*width+j+1]
      -image[(i+1)*width+j-1]-2*image[(i+1)*width+j]-image[(i+1)*width+j+1];
      s = (int) sqrtf(gx*gx+gy*gy);
      s = s > 255 ? 255:s;
      output[i*width+j] = s;
  }
}

void CheckCudaError(char sms[], int line);
float GetTime(void);

int main(int argc, char** argv) {
    CheckCudaError((char *) "First line", __LINE__);
    int width,height, pixelWidth; //meta info de la imagen
    unsigned char *image; //imagen
    unsigned int nThreads=16;
    if (argc == 1){filename = "lenna.png";}
    else if (argc == 2) {filename = argv[1];}
    else if (argc == 3) {filename = argv[1]; nThreads = atoi(argv[2]);}
    else {printf("Usage: ./cudacode.exe filename\n"); exit(0); }

    printf("Reading image...\n");
    image = stbi_load(filename, &width, &height, &pixelWidth, 1);
    if (!image) {
        fprintf(stderr, "Couldn't load image.\n");
        return (-1);
    }
    printf("Image Read. Width : %d, Height : %d, nComp: %d\n",width,height,pixelWidth);

    //Lectura feta
    hipEvent_t E1, E2;
    unsigned int numBytes = width*height*sizeof(char);
    hipEventCreate(&E1); hipEventCreate(&E2);

    
    unsigned char *imX, *imZ;
    unsigned char *image_d;
    unsigned char *image_o;
    float TiempoEle, TiempoSEQ, t1, t2;

    image_d = (unsigned char*) malloc(numBytes);

    // Ejecucion Secuencial, se ejecuta varias veces para evitar problemas de precision con el clock
    t1=GetTime();
    for (int t = 0; t<10; t++)
        sobelSeq(image, image_d, width, height);
    t2=GetTime();
    TiempoSEQ = (t2 - t1) / 10.0;

    stbi_write_png(seq_out,width,height,1,image_d,0);
    printf("Sequential image written with time %4.6f\n ms", TiempoSEQ);


  
// Pinned
    hipHostMalloc(&image_o, numBytes);
    hipHostMalloc(&image_d, numBytes);

    image_o=image;
    
    // Obtener Memoria en el device
    hipMalloc((void**)&imX, numBytes);
    hipMalloc((void**)&imZ, numBytes);
    CheckCudaError((char *) "Obtener Memoria en el device", __LINE__);

    // Copiar datos desde el host en el device
    hipMemcpy(imX, image_o, numBytes, hipMemcpyHostToDevice);
    CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);

    int nBlocksFil = (height+nThreads-1)/nThreads;
    int nBlocksCol = (width+nThreads-1)/nThreads;

    //dim3 dimGridE(Ncol/nThreads, Nfil/nThreads, 1);
    dim3 dimGridE(nBlocksCol, nBlocksFil, 1);
    dim3 dimBlockE(nThreads, nThreads, 1);

    printf("\n");
    printf("Kernel Elemento a Elemento MEMPIN\n");
    printf("Dimension problema: %d filas x %d columnas\n", height, width);
    printf("Dimension Block: %d x %d x %d (%d) threads\n", dimBlockE.x, dimBlockE.y, dimBlockE.z, dimBlockE.x * dimBlockE.y * dimBlockE.z);
    printf("Dimension Grid: %d x %d x %d (%d) blocks\n", dimGridE.x, dimGridE.y, dimGridE.z, dimGridE.x * dimGridE.y * dimGridE.z);

    hipEventRecord(E1, 0);
    hipEventSynchronize(E1);

    KernelSobelElement<<<dimGridE, dimBlockE>>>(imX, imZ, width, height);

    CheckCudaError((char *) "Invocar Kernel", __LINE__);

    hipEventRecord(E2, 0);
    hipEventSynchronize(E2);

    // Obtener el resultado desde el host
    hipMemcpy(image_d, imZ, numBytes, hipMemcpyDeviceToHost);
    CheckCudaError((char *) "Copiar Datos Device --> Host", __LINE__);

    hipFree(imX); hipFree(imZ);

    hipDeviceSynchronize();

    hipEventElapsedTime(&TiempoEle, E1, E2);

    hipEventDestroy(E1); hipEventDestroy(E2);

    stbi_write_png(cuda_out,width,height,1,image_d,0);
    printf("Image Written with CUDA execution with time: %4.6f\n ms", TiempoEle);
}

void CheckCudaError(char sms[], int line) {
  hipError_t error;

  error = hipGetLastError();
  if (error) {
    printf("(ERROR) %s - %s in %s at line %d\n", sms, hipGetErrorString(error), __FILE__, line);
    exit(EXIT_FAILURE);
  }
  //else printf("(OK) %s \n", sms);
}

float GetTime(void)        {
  struct timeval tim;
  struct rusage ru;
  getrusage(RUSAGE_SELF, &ru);
  tim=ru.ru_utime;
  return ((double)tim.tv_sec + (double)tim.tv_usec / 1000000.0)*1000.0;
}
